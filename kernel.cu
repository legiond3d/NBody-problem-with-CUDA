#include "hip/hip_runtime.h"
﻿#ifndef _Main_
#define _Main_

#include"Header.cuh"

__device__ __forceinline__ double atomicMax(double* address, double val)
{
	unsigned long long ret = __double_as_longlong(*address);
	while (val > __longlong_as_double(ret))
	{
		unsigned long long old = ret;
		if ((ret = atomicCAS((unsigned long long*)address, old, __double_as_longlong(val))) == old)
			break;
	}
	return __longlong_as_double(ret);
}

__device__ __forceinline__ float atomicMax(float* address, float val)
{
	int ret = __float_as_int(*address);
	while (val > __int_as_float(ret))
	{
		int old = ret;
		if ((ret = atomicCAS((int*)address, old, __float_as_int(val))) == old)
			break;
	}
	return __int_as_float(ret);
}

__device__ __forceinline__ double old_atomicAdd(double* address, double val)
{
	// Doing it all as longlongs cuts one __longlong_as_double from the inner loop
	unsigned long long* ptr = (unsigned long long*)address;
	unsigned long long old, newdbl, ret = *ptr;
	do {
		old = ret;
		newdbl = __double_as_longlong(__longlong_as_double(old) + val);
	} while ((ret = atomicCAS(ptr, old, newdbl)) != old);

	return __longlong_as_double(ret);
}

__device__ __forceinline__ float old_atomicAdd(float* address, float val)
{
	// Doing it all as longlongs cuts one __longlong_as_double from the inner loop
	unsigned int* ptr = (unsigned int*)address;
	unsigned int old, newint, ret = *ptr;
	do {
		old = ret;
		newint = __float_as_int(__int_as_float(old) + val);
	} while ((ret = atomicCAS(ptr, old, newint)) != old);

	return __int_as_float(ret);
}

__global__ void prep_system(const nb_vec* state_cur, nb_vec* state_new, nb_vec* K, const nb_real* k,
	const int foff, const int N, const int M, const nb_real dt)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	nb_real stateX = 0.0 , stateY = 0.0, stateZ = 0.0, stateVx = 0.0, stateVy = 0.0, stateVz = 0.0, stateM = 0.0;
	nb_real a = 0.0;

	if (i < N)
	{
		stateX = state_cur[i].x;
		stateY = state_cur[i].y;
		stateZ = state_cur[i].z;

		stateVx = state_cur[i].vx;
		stateVy = state_cur[i].vy;
		stateVz = state_cur[i].vz;

		stateM = state_cur[i].m;

		for (int j = 0; j < M; j++) //цикл по столбцам таблицы бутчера
		{
			a = k[foff + j];
			stateX += a * K[j * N + i].x * dt;
			stateY += a * K[j * N + i].y * dt;
			stateZ += a * K[j * N + i].z * dt;

			stateVx += a * K[j * N + i].vx * dt;
			stateVy += a * K[j * N + i].vy * dt;
			stateVz += a * K[j * N + i].vz * dt;
		}
		state_new[i].x = stateX;
		state_new[i].y = stateY;
		state_new[i].z = stateZ;

		state_new[i].vx = stateVx;
		state_new[i].vy = stateVy;
		state_new[i].vz = stateVz;

		state_new[i].m = stateM;
	}
}

__global__ void nb_system(const nb_vec* state_new, nb_vec* K, int foff, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = 0;

	nb_real x1 = state_new[i].x;
	nb_real y1 = state_new[i].y;
	nb_real z1 = state_new[i].z;

	nb_real vx1 = state_new[i].vx;
	nb_real vy1 = state_new[i].vy;
	nb_real vz1 = state_new[i].vz;

	nb_real dx, dy, dz, r; //координаты и модуль вектора расстояния между двумя объектами
	nb_real coef;

	nb_real local_x, local_y, local_z; //локальный результат потока

	nb_real res_x = 0.0, res_y = 0.0, res_z = 0.0; //результат потока

	extern __shared__ nb_real cache[]; //динамически выделяемая разделяемая память блока
	nb_real* x2 = cache;
	nb_real* y2 = (nb_real*)&x2[blockDim.x];
	nb_real* z2 = (nb_real*)&y2[blockDim.x];
	nb_real* m2 = (nb_real*)&z2[blockDim.x];

	for (int b = 0; b < N; b += blockDim.x) //цикл по блокам
	{
		j = b + threadIdx.x;

		if (j < N)
		{
			//копирование данных в разделяемую память блока
			x2[threadIdx.x] = state_new[j].x;
			y2[threadIdx.x] = state_new[j].y;
			z2[threadIdx.x] = state_new[j].z;
			m2[threadIdx.x] = state_new[j].m;
		}

		__syncthreads();

		local_x = 0.0;
		local_y = 0.0;
		local_z = 0.0;

		for (j = 0; j < blockDim.x; j++) //цикл по потокам внутри рассматриваемого блока
		{
			if (b + j >= N) break;
			if (i == b + j) continue;

			dx = x1 - x2[j];
			dy = y1 - y2[j];
			dz = z1 - z2[j];

			r = dx * dx + dy * dy + dz * dz;
			r = r < sigma ? sigma : r;
			coef = (G * m2[j]) / (r * sqrt(r));

			local_x -= coef * dx;
			local_y -= coef * dy;
			local_z -= coef * dz;
		}

		__syncthreads();

		res_x += local_x;
		res_y += local_y;
		res_z += local_z;
	}

	if (i < N) //сохраняем результат
	{
		i += foff;
		K[i].x = vx1;
		K[i].y = vy1;
		K[i].z = vz1;
		K[i].vx = res_x;
		K[i].vy = res_y;
		K[i].vz = res_z;
	}
}

__global__ void check_conservation(const nb_vec* state_cur, nb_real* PX, nb_real* PY, nb_real* PZ, 
	nb_real* LX, nb_real* LY, nb_real* LZ, nb_real* Energy, const int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = 0;

	nb_real x1 = state_cur[i].x;
	nb_real y1 = state_cur[i].y;
	nb_real z1 = state_cur[i].z;

	nb_real vx1 = state_cur[i].vx;
	nb_real vy1 = state_cur[i].vy;
	nb_real vz1 = state_cur[i].vz;

	nb_real m1 = state_cur[i].m;

	nb_real dx, dy, dz, r; //компоненты и модуль вектора расстояния между двумя объектами
	nb_real v; //длина (модуль) вектора скорости

	nb_real local_Ep = 0.0, Ep = 0.0, Ek = 0.0, E = 0.0; //потенциальная и кинетическая энергия объекта
	nb_real Px = 0.0, Py = 0.0, Pz = 0.0; //компоненты вектора импульса объекта
	nb_real Lx = 0.0, Ly = 0.0, Lz = 0.0; //компоненты вектора момента импульса объекта

	extern __shared__ nb_real cache[]; //динамически выделяемая разделяемая память блока
	nb_real* x2 = cache;
	nb_real* y2 = (nb_real*)&x2[blockDim.x];
	nb_real* z2 = (nb_real*)&y2[blockDim.x];
	nb_real* m2 = (nb_real*)&z2[blockDim.x];

	nb_real* e = (nb_real*)&m2[blockDim.x];
	nb_real* px = (nb_real*)&e[blockDim.x];
	nb_real* py = (nb_real*)&px[blockDim.x];
	nb_real* pz = (nb_real*)&py[blockDim.x];
	nb_real* lx = (nb_real*)&pz[blockDim.x];
	nb_real* ly = (nb_real*)&lx[blockDim.x];
	nb_real* lz = (nb_real*)&ly[blockDim.x];

	for (int b = 0; b < N; b += blockDim.x)
	{
		j = b + threadIdx.x;

		if (j < N)
		{
			//копируем данные в разделяемую память
			x2[threadIdx.x] = state_cur[j].x;
			y2[threadIdx.x] = state_cur[j].y;
			z2[threadIdx.x] = state_cur[j].z;
			m2[threadIdx.x] = state_cur[j].m;
		}

		__syncthreads();

		local_Ep = 0.0;

		for (j = 0; j < blockDim.x; j++)
		{
			if (b + j >= N) break;
			if (i == b + j) continue;
			
			dx = x1 - x2[j];
			dy = y1 - y2[j];
			dz = z1 - z2[j];

			r = dx * dx + dy * dy + dz * dz;

			r = r < sigma ? sigma : r;

			local_Ep += m2[j] / sqrt(r);
		}

		__syncthreads();

		Ep += m1 * local_Ep;
	}

	if (i < N)
	{
		v = fabs(vx1 * vx1 + vy1 * vy1 + vz1 * vz1);
		Ek = 0.5 * m1 * v;
		E = Ek + 0.5 * G * Ep;
		e[threadIdx.x] = E;

		Px = m1 * vx1;
		Py = m1 * vy1;
		Pz = m1 * vz1;
		px[threadIdx.x] = Px;
		py[threadIdx.x] = Py;
		pz[threadIdx.x] = Pz;

		Lx = m1 * (y1 * vz1 - z1 * vy1);
		Ly = m1 * (z1 * vx1 - x1 * vz1);
		Lz = m1 * (x1 * vy1 - y1 * vx1);
		lx[threadIdx.x] = Lx;
		ly[threadIdx.x] = Ly;
		lz[threadIdx.x] = Lz;
	}
	else
	{
		e[threadIdx.x] = 0.0;
		px[threadIdx.x] = 0.0;
		py[threadIdx.x] = 0.0;
		pz[threadIdx.x] = 0.0;
		lx[threadIdx.x] = 0.0;
		ly[threadIdx.x] = 0.0;
		lz[threadIdx.x] = 0.0;
	}

	__syncthreads();

	for (int idx = blockDim.x >> 1; idx > 0; idx >>= 1)
	{
		if (threadIdx.x < idx)
		{
			px[threadIdx.x] += px[threadIdx.x + idx];
			py[threadIdx.x] += py[threadIdx.x + idx];
			pz[threadIdx.x] += pz[threadIdx.x + idx];

			lx[threadIdx.x] += lx[threadIdx.x + idx];
			ly[threadIdx.x] += ly[threadIdx.x + idx];
			lz[threadIdx.x] += lz[threadIdx.x + idx];

			e[threadIdx.x] += e[threadIdx.x + idx];
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) //сумма всех элементов каждого блока находится в первом потоке
	{
		//используются атомарные операции сложения для записи в глобальную память суммы результатов каждого блока
		old_atomicAdd(PX, px[0]);
		old_atomicAdd(PY, py[0]);
		old_atomicAdd(PZ, pz[0]);
		old_atomicAdd(LX, lx[0]);
		old_atomicAdd(LY, ly[0]);
		old_atomicAdd(LZ, lz[0]);
		old_atomicAdd(Energy, e[0]);
	}
}

__global__ void detect_err(nb_vec* K, const nb_real* b1, const nb_real* b2, const int N, 
	const int M, const nb_real dt, nb_real* E) //функция для оценки ошибки, полученной вложенным методом
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int idx = threadIdx.x;

	extern __shared__ nb_real cache[];
	nb_real* Err = cache;

	nb_real ErrX = 0.0, ErrY = 0.0, ErrZ = 0.0, ErrVx = 0.0, ErrVy = 0.0, ErrVz = 0.0;
	nb_real err = 0.0, max_err = 0.0;
	nb_real errL = 0.0, errR = 0.0;

	if (i < N)
	{
		for (int j = 0; j < M; j++)
		{
			ErrX += dt * (b1[j] - b2[j]) * K[j * N + i].x;
			ErrY += dt * (b1[j] - b2[j]) * K[j * N + i].y;
			ErrZ += dt * (b1[j] - b2[j]) * K[j * N + i].z;

			ErrVx += dt * (b1[j] - b2[j]) * K[j * N + i].vx;
			ErrVy += dt * (b1[j] - b2[j]) * K[j * N + i].vy;
			ErrVz += dt * (b1[j] - b2[j]) * K[j * N + i].vz;
		}
		err = fabs(ErrX);
		max_err = max_err < err ? err : max_err;
		err = fabs(ErrY);
		max_err = max_err < err ? err : max_err;
		err = fabs(ErrZ);
		max_err = max_err < err ? err : max_err;

		err = fabs(ErrVx);
		max_err = max_err < err ? err : max_err;
		err = fabs(ErrVy);
		max_err = max_err < err ? err : max_err;
		err = fabs(ErrVz);
		max_err = max_err < err ? err : max_err;
	}
	Err[threadIdx.x] = i < N ? max_err : 0.0;
	__syncthreads();
	for (idx = blockDim.x >> 1; idx > 0; idx >>= 1)
	{
		if (threadIdx.x < idx)
		{
			errL = Err[threadIdx.x];
			errR = Err[threadIdx.x + idx];
			Err[threadIdx.x] = errL < errR ? errR : errL;
		}
		__syncthreads();
 	}
	if (threadIdx.x == 0) //в первом потоке каждого блока находится максимальное значение погрешности
	{
		atomicMax(E, Err[0]); //с помощью атомарной операции сравнения записываем результат в глобальную память
	}
}

__global__ void solve_system(const nb_vec* state_cur, nb_vec* state_new, nb_vec* K, const nb_real* b1,
	const int N, const int M, const nb_real dt)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	nb_real stateX, stateY, stateZ, stateVx, stateVy, stateVz;

	if (i < N)
	{
		stateX = state_cur[i].x;
		stateY = state_cur[i].y;
		stateZ = state_cur[i].z;

		stateVx = state_cur[i].vx;
		stateVy = state_cur[i].vy;
		stateVz = state_cur[i].vz;
		
		for (int j = 0; j < M; j++)
		{
			stateX += dt * b1[j] * K[j * N + i].x;
			stateY += dt * b1[j] * K[j * N + i].y;
			stateZ += dt * b1[j] * K[j * N + i].z;

			stateVx += dt * b1[j] * K[j * N + i].vx;
			stateVy += dt * b1[j] * K[j * N + i].vy;
			stateVz += dt * b1[j] * K[j * N + i].vz;
		}

		state_new[i].x = stateX;
		state_new[i].y = stateY;
		state_new[i].z = stateZ;

		state_new[i].vx = stateVx;
		state_new[i].vy = stateVy;
		state_new[i].vz = stateVz;
		
		//используем свойство FSAL
		K[i].x = K[(M - 1) * N + i].x;
		K[i].y = K[(M - 1) * N + i].y;
		K[i].z = K[(M - 1) * N + i].z;

		K[i].vx = K[(M - 1) * N + i].vx;
		K[i].vy = K[(M - 1) * N + i].vy;
		K[i].vz = K[(M - 1) * N + i].vz;
	}
}

__host__ void nb_check_conservation_law(const nb_vec* state_cur, nb_real* E, nb_real* PX, nb_real* PY, nb_real* PZ,
	nb_real* LX, nb_real* LY, nb_real* LZ, nb_real& e, nb_real& px, nb_real& py, nb_real& pz, 
	nb_real& lx, nb_real& ly, nb_real& lz, const int& N)
{
	int block = BLOCK_SIZE; //размер блока
	int grid = N % block == 0 ? N / block : N / block + 1; //размер решётки блоков (кол-во блоков)

	e = px = py = pz = lx = ly = lz = 0.0;

	hipMemcpy(E, &e, sizeof(nb_real), hipMemcpyHostToDevice);

	hipMemcpy(PX, &px, sizeof(nb_real), hipMemcpyHostToDevice);
	hipMemcpy(PY, &py, sizeof(nb_real), hipMemcpyHostToDevice);
	hipMemcpy(PZ, &pz, sizeof(nb_real), hipMemcpyHostToDevice);

	hipMemcpy(LX, &lx, sizeof(nb_real), hipMemcpyHostToDevice);
	hipMemcpy(LY, &ly, sizeof(nb_real), hipMemcpyHostToDevice);
	hipMemcpy(LZ, &lz, sizeof(nb_real), hipMemcpyHostToDevice);

	check_conservation <<< grid, block, 11 * block * sizeof(nb_real) >>> (state_cur, PX, PY, PZ, LX, LY, LZ, E, N);

	hipMemcpy(&e, E, sizeof(nb_real), hipMemcpyDeviceToHost);

	hipMemcpy(&px, PX, sizeof(nb_real), hipMemcpyDeviceToHost);
	hipMemcpy(&py, PY, sizeof(nb_real), hipMemcpyDeviceToHost);
	hipMemcpy(&pz, PZ, sizeof(nb_real), hipMemcpyDeviceToHost);

	hipMemcpy(&lx, LX, sizeof(nb_real), hipMemcpyDeviceToHost);
	hipMemcpy(&ly, LY, sizeof(nb_real), hipMemcpyDeviceToHost);
	hipMemcpy(&lz, LZ, sizeof(nb_real), hipMemcpyDeviceToHost);
}

__host__ nb_real nb_solver(const nb_vec* state_cur, nb_vec* state_new, nb_vec* K, const nb_real* a, 
	const nb_real* b1, const nb_real* b2, const nb_real* c, const bool& inclose, const bool& inplicit,
	bool& accept, const bool& first, const int& N, const int& M, const int& order, nb_real* Err, const nb_real& dt)
{
	int block = BLOCK_SIZE; //размер блока
	int grid = N % block == 0 ? N / block : N / block + 1; //размер решётки блоков (кол-во блоков)

	nb_real dt_new = dt;
	nb_real scale = 1.0;

	for (int kk = first ? 0 : 1; kk < M; kk++)
	{
		prep_system <<< grid, block >>> (state_cur, state_new, K, a, kk * M, N, inplicit ? M : kk, dt);
		nb_system <<< grid, block, 4 * block * sizeof(nb_real) >>> (state_new, K, kk * N, N);
	}
	if (inclose) //если метод вложенный применяется оценка погрешности
	{
		nb_real err = 0.0;
		hipMemcpy(Err, &err, sizeof(nb_real), hipMemcpyHostToDevice);
		detect_err <<< grid, block, block * sizeof(nb_real) >>> (K, b1, b2, N, M, dt, Err);
		hipMemcpy(&err, Err, sizeof(nb_real), hipMemcpyDeviceToHost);
		accept = err <= threshold ? true : false; //принимаем решение о принятии шага
		scale = 0.8 * pow(threshold / err, 1.0 / order);
		scale = min(max(min_step, scale), max_step);
		dt_new = scale * dt;
	}
	if (accept) //если решено шаг принять
		solve_system <<< grid, block >>> (state_cur, state_new, K, b1, N, M, dt);
	return dt_new;
}

int main()
{
	setlocale(LC_ALL, "Rus");
	//объявление переменных
	int N; //число объектов
	double t = 0.0; //текущее время (время начала моделирования)
	double end = 10.0;//365.2 * 24.0 * 60.0 * 60.0; //время конца моделирования (в секундах)
	double dt = 0.005; //начальный шаг по времени (в секундах)
	double dt_new = dt; //новый шаг по времени
	double tsave = 15.0; //время промежуточного сохранения в файл
	double tcheck = 0.1;
	double save = tsave; //таймер для сохранения в файл
	double check = tcheck;
	bool accept = true; //флаг, определяющий будет ли принят временной шаг или отклонён
	bool swap = true; //флаг, позволяющий поменять местами state_cur и state_new
	bool first = true; //флаг, определяющий первый шаг (нужен для реализации свойства FSAL)

	nb_vec* state_cur; //указатель на массив структуры для хранения текущего состояния системы
	nb_vec* K; //указатель на двумерный массив для хранения промежуточных состояний системы в методе РК
	rkdp solver; //объявление класса решателя методом РК
	const int M = solver.Steps(); //количество стадий метода РК
	const int order = solver.Order(); //наивысший порядок метода РК
	const bool inplicit = solver.inplicit(); //является ли метод РК неявным
	const bool inclose = solver.inclose(); //является ли метод РК вложенным
	const nb_real** a = solver.A(); //коэффициенты правой части таблицы Бутчера
	nb_real* aa;
	const nb_real* c = solver.C(); //коэффициенты левой части
	const nb_real* b1 = solver.B1(); //коэффициенты для основного решения
	const nb_real* b2 = solver.B2(); //коэффициенты для вспомогательного решения

	nb_real e0 = 0.0, px0 = 0.0, py0 = 0.0, pz0 = 0.0, lx0 = 0.0, ly0 = 0.0, lz0 = 0.0; //начальная энергия, импульс и момент импульса системы
	nb_real e, px, py, pz, lx, ly, lz;
	nb_real de, dp, dpx, dpy, dpz, dl, dlx, dly, dlz;

	ifstream file_input("test2.txt"); //открытие файла для чтения
	ofstream file_output("result.txt"); //открыие файла для записи
	ofstream file_conserv("consetvation.txt");
	file_input >> N; //чтение количества объектов моделирования

	cout << "Начальные условия:" << endl << endl;
	cout << "N = " << N << ", t0 = " << t << ", t_end = " << end << ", BLOCK_SIZE = " << BLOCK_SIZE << endl << endl;

	//выделение памяти
	state_cur = new nb_vec[N];
	K = new nb_vec[M * N];
	aa = new nb_real[M * M];

	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < M; j++)
		{
			aa[i * M + j] = a[i][j];
		}
	}

	//инициализация начальных условий
	for (int i = 0; i < N; i++)
	{
		//чтение данных из файла:
		file_input >> state_cur[i].x >> state_cur[i].y >> state_cur[i].z;
		file_input >> state_cur[i].vx >> state_cur[i].vy >> state_cur[i].vz;
		file_input >> state_cur[i].m;

		if (N < 50)
		{
			cout << "r[" << i << "] = { " << state_cur[i].x << ", " << state_cur[i].y << ", " << state_cur[i].z << " }" << endl;
			cout << "v[" << i << "] = { " << state_cur[i].vx << ", " << state_cur[i].vy << ", " << state_cur[i].vz << " }" << endl;
			cout << "m[" << i << "] = { " << state_cur[i].m << " }" << endl << endl;
		}

		if (inplicit) //если метод является неявным, необходимо инициализировать начальное приближение
			for (int j = 0; j < M; j++)
			{
				K[j * N + i].x = state_cur[i].x;
				K[j * N + i].y = state_cur[i].y;
				K[j * N + i].z = state_cur[i].z;

				K[j * N + i].vx = state_cur[i].vx;
				K[j * N + i].vy = state_cur[i].vy;
				K[j * N + i].vz = state_cur[i].vz;

				K[j * N + i].m = state_cur[i].m;
			}
	}
	file_input.close();

	//GPU:
	nb_vec* State_cur, * State_new; //указатели на массивы структур для хранения текущего и нового состояния системы
	nb_vec* KK; //указатель на двумерный массив для хранения промежуточных состояний системы в методе РК
	nb_real* A, * B1, * B2, * C;
	nb_real* Err = nullptr, * E, * Px, * Py, * Pz, * Lx, * Ly, * Lz;
	hipSetDevice(0);
	{//выделение памяти под массивы на устройстве
		hipMalloc((void**)&Err, sizeof(nb_real));
		hipMalloc((void**)&E, sizeof(nb_real));
		hipMalloc((void**)&Px, sizeof(nb_real));
		hipMalloc((void**)&Py, sizeof(nb_real));
		hipMalloc((void**)&Pz, sizeof(nb_real));
		hipMalloc((void**)&Lx, sizeof(nb_real));
		hipMalloc((void**)&Ly, sizeof(nb_real));
		hipMalloc((void**)&Lz, sizeof(nb_real));
		hipMalloc((void**)&State_cur, N * sizeof(nb_vec));
		hipMalloc((void**)&State_new, N * sizeof(nb_vec));
		hipMalloc((void**)&KK, N * M * sizeof(nb_vec));
		hipMalloc((void**)&A, M * M * sizeof(nb_real));
		hipMalloc((void**)&B1, M * sizeof(nb_real));
		hipMalloc((void**)&B2, M * sizeof(nb_real));
		hipMalloc((void**)&C, M * sizeof(nb_real));
	}
	{//копирование массивов на устройство
		hipMemcpy(State_cur, state_cur, N * sizeof(nb_vec), hipMemcpyHostToDevice);
		hipMemcpy(State_new, state_cur, N * sizeof(nb_vec), hipMemcpyHostToDevice);
		if (inplicit)
			hipMemcpy(KK, K, N * M * sizeof(nb_vec), hipMemcpyHostToDevice);
		hipMemcpy(A, aa, M * M * sizeof(nb_real), hipMemcpyHostToDevice);
		hipMemcpy(B1, b1, M * sizeof(nb_real), hipMemcpyHostToDevice);
		hipMemcpy(B2, b2, M * sizeof(nb_real), hipMemcpyHostToDevice);
		hipMemcpy(C, c, M * sizeof(nb_real), hipMemcpyHostToDevice);
	}

	nb_check_conservation_law(State_cur, E, Px, Py, Pz, Lx, Ly, Lz, e0, px0, py0, pz0, lx0, ly0, lz0, N);
	cout << endl << "E0 = " << e0 << ", P0 = { " << px0 << ", " << py0 << ", " << pz0 << " }, L0 = { " << lx0 << ", " << ly0 << ", " << lz0 << " }" << endl << endl;

	clock_t start = clock();

	//основной цикл по времени
	while (t < end)
	{
		if (swap)
			dt_new = nb_solver(State_cur, State_new, KK, A, B1, B2, C, inclose, inplicit, accept, first, N, M, order, Err, dt);
		else
			dt_new = nb_solver(State_new, State_cur, KK, A, B1, B2, C, inclose, inplicit, accept, first, N, M, order, Err, dt);
		if (accept) //если шаг по времени был принят
		{
			first = false;
			swap = !swap; //меняем местами state_new и state_cur
			t += dt; //делаем шаг по времени
			cout << "dt = " << dt << ", t = " << t << ", end = " << end << ", swap = " << swap << ", затрачено времени = " << (clock() - start) / 1000.0 << " c." << endl;
			start = clock();
			if (int(t / save) == 1) //сохраняем промежуточное состояние системы
			{
				if (swap)
					hipMemcpy(state_cur, State_cur, N * sizeof(nb_vec), hipMemcpyDeviceToHost);
				else
					hipMemcpy(state_cur, State_new, N * sizeof(nb_vec), hipMemcpyDeviceToHost);
				for (int i = 0; i < N; i++)
				{
					file_output << state_cur[i].x << "," << state_cur[i].y << "," << state_cur[i].z << ","
					<< state_cur[i].vx << "," << state_cur[i].vy << "," << state_cur[i].vz << ","
					<< state_cur[i].m << endl;
				}
				save += tsave;
			}
			//if (int(t / check) == 1)
			{
				if (swap)
					nb_check_conservation_law(State_cur, E, Px, Py, Pz, Lx, Ly, Lz, e, px, py, pz, lx, ly, lz, N);
				else
					nb_check_conservation_law(State_new, E, Px, Py, Pz, Lx, Ly, Lz, e, px, py, pz, lx, ly, lz, N);

				de = fabs((e - e0) / e0);

				dpx = (px - px0) / px0;
				dpy = (py - py0) / py0;
				dpz = (pz - pz0) / pz0;
				dp = sqrt(dpx * dpx + dpy * dpy + dpz * dpz);

				dlx = (lx - lx0) / lx0;
				dly = (ly - ly0) / ly0;
				dlz = (lz - lz0) / lz0;
				dl = sqrt(dlx * dlx + dly * dly + dlz * dlz);

				file_conserv << t << "," << de << "," << dp << "," << dl << endl;

				//check += tcheck;
			}
		}
		dt = dt_new <= end - t ? dt_new : end - t; //шаг по времени для следующего шага
	}

	if (swap)
		hipMemcpy(state_cur, State_cur, N * sizeof(nb_vec), hipMemcpyDeviceToHost);
	else
		hipMemcpy(state_cur, State_new, N * sizeof(nb_vec), hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++)
	{
		file_output << state_cur[i].x << "," << state_cur[i].y << "," << state_cur[i].z << ","
			<< state_cur[i].vx << "," << state_cur[i].vy << "," << state_cur[i].vz << ","
			<< state_cur[i].m << "," << sqrt(state_cur[i].vx * state_cur[i].vx + state_cur[i].vy * state_cur[i].vy + state_cur[i].vz * state_cur[i].vz) << endl;
	}
	
	file_output.close();
	file_conserv.close();
	//освобождение памяти
	delete[] state_cur;
	delete[] K;
	delete[] aa;

	hipFree(State_cur);
	hipFree(State_new);
	hipFree(KK);
	hipFree(A);
	hipFree(B1);
	hipFree(B2);
	hipFree(C);
	hipFree(Err);
	hipFree(E);
	hipFree(Px);
	hipFree(Py);
	hipFree(Pz);
	hipFree(Lx);
	hipFree(Ly);
	hipFree(Lz);

	//system("pause");
	return 0;
}

#endif